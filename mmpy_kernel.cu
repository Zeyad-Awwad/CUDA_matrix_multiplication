#include "hip/hip_runtime.h"
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "utils.h"
#include "types.h"

#define TW 32

using namespace std;

__global__ void matMul(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B)
{

    __shared__ double As[TW][TW], Bs[TW][TW];
    
    int ty = threadIdx.y;
    int tx = threadIdx.x;
    int I = blockIdx.y * TW + 8*ty;
    int J =  blockIdx.x * TW + 2*tx;
    int border = N - I;
    if (border > 8)
        border = 8;
    int kk;
    
    if((I < N) && (J < N))
    {
        double c00, c01, c10, c11, c20, c30, c21, c31;
        double c40, c50, c41, c51, c60, c70, c61, c71;
        c00 = 0; c01 = 0; c10 = 0; c11 = 0;
        c20 = 0; c30 = 0; c21 = 0; c31 = 0;
        c40 = 0; c50 = 0; c41 = 0; c51 = 0;
        c60 = 0; c70 = 0; c61 = 0; c71 = 0;
        
        for (kk=0; kk<N/TW; kk++) // go through each block
        {
            // read each block into shared memory
            
            #pragma unroll
            for (int i=0; i<border; i++)
            {
                As[8*ty+i][2*tx] = A[(I+i)*N + kk*TW + 2*tx];
                As[8*ty+i][2*tx+1] = A[(I+i)*N + kk*TW + 2*tx + 1];
            }
            #pragma unroll
            for (int i=0; i<border; i++)
            { 
                Bs[8*ty+i][2*tx] = B[(kk*TW + 8*ty+i)*N + J];
                Bs[8*ty+i][2*tx+1] = B[(kk*TW + 8*ty+i)*N + J + 1];
            }
            __syncthreads();
            
            // dot product
            #pragma unroll
            for (int k=0; k < TW; k++)
            {
                c00 += As[8*ty][k] * Bs[k][2*tx];
                c01 += As[8*ty][k] * Bs[k][2*tx+1];
                c10 += As[8*ty+1][k] * Bs[k][2*tx];
                c11 += As[8*ty+1][k] * Bs[k][2*tx+1];
                c20 += As[8*ty+2][k] * Bs[k][2*tx];
                c21 += As[8*ty+2][k] * Bs[k][2*tx+1];
                c30 += As[8*ty+3][k] * Bs[k][2*tx];
                c31 += As[8*ty+3][k] * Bs[k][2*tx+1];
                c40 += As[8*ty+4][k] * Bs[k][2*tx];
                c41 += As[8*ty+4][k] * Bs[k][2*tx+1];
                c50 += As[8*ty+5][k] * Bs[k][2*tx];
                c51 += As[8*ty+5][k] * Bs[k][2*tx+1];
                c60 += As[8*ty+6][k] * Bs[k][2*tx];
                c61 += As[8*ty+6][k] * Bs[k][2*tx+1];
                c70 += As[8*ty+7][k] * Bs[k][2*tx];
                c71 += As[8*ty+7][k] * Bs[k][2*tx+1];
            }
            __syncthreads();
        }
        ///*
        int diff = N - kk*TW;
        if ( diff > 0 )
        {   
            for (int i=0; i<border; i++)
            {
                As[8*ty+i][2*tx] = 0;
                As[8*ty+i][2*tx+1] = 0;
                Bs[8*ty+i][2*tx] = 0;
                Bs[8*ty+i][2*tx+1] = 0;
            }
            
            if (2*tx < diff)
            {
                for (int i=0; i<border; i++)
                {
                    As[8*ty+i][2*tx] = A[(I+i)*N + kk*TW + 2*tx];
                }
            }
            if (2*tx + 1 < diff)
            {
                for (int i=0; i<border; i++)
                {
                    As[8*ty+i][2*tx+1] = A[(I+i)*N + kk*TW + 2*tx + 1];
                }
            }
            
            for (int i=0; i<border; i++)
            {
                if (8*ty + i < diff + 2 )
                {
                    Bs[8*ty+i][2*tx] = B[(kk*TW + 8*ty+i)*N + J];
                }
            }
            if ( (J+1) < N)
            {
                for (int i=0; i<border; i++)
                {
                    if (8*ty + i < diff + 2)
                    {
                        Bs[8*ty+i][2*tx+1] = B[(kk*TW + 8*ty+i)*N + J+1];
                    }
                }
            }
            
            __syncthreads();

            // dot product
            #pragma unroll
            for (int k=0; k < TW; k++)
            {
                c00 += As[8*ty][k] * Bs[k][2*tx];
                c01 += As[8*ty][k] * Bs[k][2*tx+1];
                c10 += As[8*ty+1][k] * Bs[k][2*tx];
                c11 += As[8*ty+1][k] * Bs[k][2*tx+1];
                c20 += As[8*ty+2][k] * Bs[k][2*tx];
                c21 += As[8*ty+2][k] * Bs[k][2*tx+1];
                c30 += As[8*ty+3][k] * Bs[k][2*tx];
                c31 += As[8*ty+3][k] * Bs[k][2*tx+1];
                c40 += As[8*ty+4][k] * Bs[k][2*tx];
                c41 += As[8*ty+4][k] * Bs[k][2*tx+1];
                c50 += As[8*ty+5][k] * Bs[k][2*tx];
                c51 += As[8*ty+5][k] * Bs[k][2*tx+1];
                c60 += As[8*ty+6][k] * Bs[k][2*tx];
                c61 += As[8*ty+6][k] * Bs[k][2*tx+1];
                c70 += As[8*ty+7][k] * Bs[k][2*tx];
                c71 += As[8*ty+7][k] * Bs[k][2*tx+1];
            }
            
            __syncthreads();
        }
        //*/
        
        C[I*N + J] = c00;
        if ( (I+1) < N )
            C[(I+1)*N + J] = c10;
        if ( (I+2) < N )
            C[(I+2)*N + J] = c20;
        if ( (I+3) < N )
            C[(I+3)*N + J] = c30;
        if ( (I+4) < N )
            C[(I+4)*N + J] = c40;
        if ( (I+5) < N )
            C[(I+5)*N + J] = c50;
        if ( (I+6) < N )
            C[(I+6)*N + J] = c60;
        if ( (I+7) < N )
            C[(I+7)*N + J] = c70;
        if ( (J+1) < N )
        {
            C[I*N + J + 1] = c01;
            if ( (I+1) < N )
                C[(I+1)*N + J + 1] = c11;
            if ( (I+2) < N )
                C[(I+2)*N + J + 1] = c21;
            if ( (I+3) < N )
                C[(I+3)*N + J + 1] = c31;
            if ( (I+4) < N )
                C[(I+4)*N + J + 1] = c41;
            if ( (I+5) < N )
                C[(I+5)*N + J + 1] = c51;
            if ( (I+6) < N )
                C[(I+6)*N + J + 1] = c61;
            if ( (I+7) < N )
                C[(I+7)*N + J + 1] = c71;
        }
    }
}